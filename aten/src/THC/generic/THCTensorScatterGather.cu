
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorScatterGather.cu"
#else

#define RUN(TYPE, DIMS, REAL)                                           \
  THCudaTensor_gatherKernel<TYPE, REAL, DIMS>                                \
  <<<grid, block, 0, THCState_getCurrentStreamOnDevice(state, curDevice)>>>(               \
    tensorInfo, srcInfo, indexInfo, dim, (TYPE)totalElements);

void THCTensor_(gather)(THCState* state, THCTensor *tensor,
                         THCTensor *src, int dim, THCudaLongTensor *index) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, tensor, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  THArgCheck(THCudaLongTensor_nDimensionLegacyNoScalars(state, index) == THCTensor_(nDimensionLegacyNoScalars)(state, src), 4,
             "Index tensor must have same dimensions as input tensor");
  THArgCheck(tensor->sizes().equals(index->sizes()), 4,
             "Index tensor must have the same size as output tensor.");
  THArgCheck(dim >= 0 && dim < THCTensor_(nDimensionLegacyNoScalars)(state, tensor), 3,
             "Index dimension is out of bounds");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, src) == THCTensor_(nDimensionLegacyNoScalars)(state, tensor), 2,
             "Input tensor must have same dimensions as output tensor");

  for (int d = 0; d < THCTensor_(nDimensionLegacyNoScalars)(state, tensor); d++) {
    if (d != dim) {
      THArgCheck(THCTensor_(sizeLegacyNoScalars)(state, tensor, d) == THCTensor_(sizeLegacyNoScalars)(state, src, d), 2,
                 "Input tensor must have same size as output tensor apart from the specified dimension");
    }
  }

  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, tensor) <= MAX_CUTORCH_DIMS,
             1, CUTORCH_DIM_WARNING);


  const ptrdiff_t totalElements = THCudaLongTensor_nElement(state, index);
  const dim3 block = getApplyBlock();
  dim3 grid;
  int curDevice = -1;
  cudaGetDevice(&curDevice);
  THArgCheck(getApplyGrid(state, totalElements, grid, curDevice), 1, CUTORCH_DIM_WARNING);

  THCTensor* oldTensor = NULL;
  if (THCTensor_maybeOverlappingIndices(state, tensor)) {
    oldTensor = tensor;
    tensor = THCTensor_(newContiguous)(state, tensor);
  }

  if (totalElements > 0) {
    if (THCTensor_canUse32BitIndexMath(state, tensor) &&
        THCTensor_canUse32BitIndexMath(state, src) &&
        THCTensor_canUse32BitIndexMath(state, index)) {
      TensorInfo<scalar_t, unsigned int> tensorInfo =
        getTensorInfo<scalar_t, THCTensor, unsigned int>(state, tensor);
      TensorInfo<scalar_t, unsigned int> srcInfo =
        getTensorInfo<scalar_t, THCTensor, unsigned int>(state, src);
      TensorInfo<int64_t, unsigned int> indexInfo =
        getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, index);

      // Specialize for a small number of dimensions.
      switch (indexInfo.dims) {
        case 1:
          RUN(unsigned int, 1, scalar_t);
          THCudaCheck(cudaGetLastError());
          break;
        case 2:
          RUN(unsigned int, 2, scalar_t);
          THCudaCheck(cudaGetLastError());
          break;
        case 3:
          RUN(unsigned int, 3, scalar_t);
          THCudaCheck(cudaGetLastError());
          break;
        default:
          RUN(unsigned int, -1, scalar_t);
          THCudaCheck(cudaGetLastError());
          break;
      }
    } else {
      TensorInfo<scalar_t, uint64_t> tensorInfo =
        getTensorInfo<scalar_t, THCTensor, uint64_t>(state, tensor);
      TensorInfo<scalar_t, uint64_t> srcInfo =
        getTensorInfo<scalar_t, THCTensor, uint64_t>(state, src);
      TensorInfo<int64_t, uint64_t> indexInfo =
        getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, index);
      RUN(uint64_t, -1, scalar_t);
      THCudaCheck(cudaGetLastError());
    }
  }

  if (oldTensor) {
    THCTensor_copyIgnoringOverlaps<scalar_t>(state, oldTensor, tensor);
    THCTensor_(free)(state, tensor);
    tensor = oldTensor;
  }
  THCudaCheck(cudaGetLastError());
}

#undef RUN


#define RUN(TYPE, DIMS, REAL)                                           \
  THCudaTensor_scatterKernel<TYPE, REAL, DIMS>                               \
  <<<grid, block, 0, THCState_getCurrentStreamOnDevice(state, curDevice)>>>(               \
    tensorInfo, srcInfo, indexInfo, dim, (TYPE)totalElements);

void THCTensor_(scatter)(THCState* state, THCTensor *tensor, int dim, THCudaLongTensor *index, THCTensor *src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, tensor, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  int index_ndim_legacy_all = THCudaLongTensor_nDimensionLegacyAll(state, index);
  THArgCheck(dim >= 0 && dim < THCTensor_(nDimensionLegacyNoScalars)(state, tensor), 2,
             "Index dimension is out of bounds");
  THArgCheck(index_ndim_legacy_all == 0
             || THCudaLongTensor_nDimensionLegacyNoScalars(state, index) == THCTensor_(nDimensionLegacyNoScalars)(state, src), 3,
             "Index tensor must be either empty or have same dimensions as input tensor");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, src) == THCTensor_(nDimensionLegacyNoScalars)(state, tensor), 4,
             "Input tensor must have same dimensions as output tensor");

  // no-op if index is empty
  if (index_ndim_legacy_all == 0)
      return;

  for (int d = 0; d < THCTensor_(nDimensionLegacyNoScalars)(state, tensor); d++) {
    int64_t indexSizeD = THCudaLongTensor_sizeLegacyNoScalars(state, index, d);
    if (d != dim) {
      THArgCheck(indexSizeD <= THCTensor_(sizeLegacyNoScalars)(state, tensor, d), 3,
                 "Index tensor must not have larger size than output tensor apart from the specified dimension %d, but got index %s output %s",
                 dim, THCudaLongTensor_sizeDesc(state, index).str, THCTensor_(sizeDesc)(state, tensor).str);
    }
    THArgCheck(indexSizeD <= THCTensor_(sizeLegacyNoScalars)(state, src, d), 3,
               "Index tensor must not have larger size than input tensor, but got index %s input %s",
               THCudaLongTensor_sizeDesc(state, index).str, THCTensor_(sizeDesc)(state, src).str);
  }

  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, tensor) <= MAX_CUTORCH_DIMS,
             1, CUTORCH_DIM_WARNING);

  const ptrdiff_t totalElements = THCudaLongTensor_nElement(state, index);
  const dim3 block = getApplyBlock();
  dim3 grid;
  int curDevice = -1;
  cudaGetDevice(&curDevice);
  THArgCheck(getApplyGrid(state, totalElements, grid, curDevice), 1, CUTORCH_DIM_WARNING);

  THCTensor* oldTensor = NULL;
  if (THCTensor_maybeOverlappingIndices(state, tensor)) {
    oldTensor = tensor;
    tensor = THCTensor_(newContiguous)(state, tensor);
  }

  if (totalElements > 0) {
    if (THCTensor_canUse32BitIndexMath(state, tensor) &&
        THCTensor_canUse32BitIndexMath(state, src) &&
        THCTensor_canUse32BitIndexMath(state, index)) {
      TensorInfo<scalar_t, unsigned int> tensorInfo =
        getTensorInfo<scalar_t, THCTensor, unsigned int>(state, tensor);
      TensorInfo<scalar_t, unsigned int> srcInfo =
        getTensorInfo<scalar_t, THCTensor, unsigned int>(state, src);
      TensorInfo<int64_t, unsigned int> indexInfo =
        getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, index);

      // Specialize for a small number of dimensions.
      switch (indexInfo.dims) {
        case 1:
          RUN(unsigned int, 1, scalar_t);
          break;
        case 2:
          RUN(unsigned int, 2, scalar_t);
          break;
        case 3:
          RUN(unsigned int, 3, scalar_t);
          break;
        default:
          RUN(unsigned int, -1, scalar_t);
          break;
      }
    } else {
      TensorInfo<scalar_t, uint64_t> tensorInfo =
        getTensorInfo<scalar_t, THCTensor, uint64_t>(state, tensor);
      TensorInfo<scalar_t, uint64_t> srcInfo =
        getTensorInfo<scalar_t, THCTensor, uint64_t>(state, src);
      TensorInfo<int64_t, uint64_t> indexInfo =
        getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, index);

      RUN(uint64_t, -1, scalar_t)
    }
  }

  if (oldTensor) {
    THCTensor_copyIgnoringOverlaps<scalar_t>(state, oldTensor, tensor);
    THCTensor_(free)(state, tensor);
    tensor = oldTensor;
  }
  THCudaCheck(cudaGetLastError());
}

#undef RUN

#define RUN(TYPE, DIMS, REAL)                                           \
  THCudaTensor_scatterAddKernel<TYPE, REAL, DIMS>                               \
  <<<grid, block, 0, THCState_getCurrentStreamOnDevice(state, curDevice)>>>(               \
    tensorInfo, srcInfo, indexInfo, dim, (TYPE)totalElements);

void THCTensor_(scatterAdd)(THCState* state, THCTensor *tensor, int dim, THCudaLongTensor *index, THCTensor *src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, tensor, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  THArgCheck(dim >= 0 && dim < THCTensor_(nDimensionLegacyNoScalars)(state, tensor), 2,
             "Index dimension is out of bounds");
  int index_ndim_legacy_all = THCudaLongTensor_nDimensionLegacyAll(state, index);

  THArgCheck(index_ndim_legacy_all == 0
             || THCudaLongTensor_nDimensionLegacyNoScalars(state, index) == THCTensor_(nDimensionLegacyNoScalars)(state, src), 3,
             "Index tensor must either be empty or have same dimensions as input tensor");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, src) == THCTensor_(nDimensionLegacyNoScalars)(state, tensor), 4,
             "Input tensor must have same dimensions as output tensor");

  // no-op if index is empty
  if (index_ndim_legacy_all == 0)
      return;

  for (int d = 0; d < THCTensor_(nDimensionLegacyNoScalars)(state, tensor); d++) {
    int64_t indexSizeD = THCudaLongTensor_sizeLegacyNoScalars(state, index, d);
    if (d != dim) {
      THArgCheck(indexSizeD <= THCTensor_(sizeLegacyNoScalars)(state, tensor, d), 3,
                 "Index tensor must not have larger size than output tensor apart from the specified dimension %d, but got index %s output %s",
                 dim, THCudaLongTensor_sizeDesc(state, index).str, THCTensor_(sizeDesc)(state, tensor).str);
    }
    THArgCheck(indexSizeD <= THCTensor_(sizeLegacyNoScalars)(state, src, d), 3,
               "Index tensor must not have larger size than input tensor, but got index %s input %s",
               THCudaLongTensor_sizeDesc(state, index).str, THCTensor_(sizeDesc)(state, src).str);
  }

  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, tensor) <= MAX_CUTORCH_DIMS,
             1, CUTORCH_DIM_WARNING);

  const ptrdiff_t totalElements = THCudaLongTensor_nElement(state, index);
  const dim3 block = getApplyBlock();
  dim3 grid;
  int curDevice = -1;
  cudaGetDevice(&curDevice);
  THArgCheck(getApplyGrid(state, totalElements, grid, curDevice), 1, CUTORCH_DIM_WARNING);

  THCTensor* oldTensor = NULL;
  if (THCTensor_maybeOverlappingIndices(state, tensor)) {
    oldTensor = tensor;
    tensor = THCTensor_(newContiguous)(state, tensor);
  }

  if (totalElements > 0) {
    if (THCTensor_canUse32BitIndexMath(state, tensor) &&
        THCTensor_canUse32BitIndexMath(state, src) &&
        THCTensor_canUse32BitIndexMath(state, index)) {
      TensorInfo<scalar_t, unsigned int> tensorInfo =
        getTensorInfo<scalar_t, THCTensor, unsigned int>(state, tensor);
      TensorInfo<scalar_t, unsigned int> srcInfo =
        getTensorInfo<scalar_t, THCTensor, unsigned int>(state, src);
      TensorInfo<int64_t, unsigned int> indexInfo =
        getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, index);

      // Specialize for a small number of dimensions.
      switch (indexInfo.dims) {
        case 1:
          RUN(unsigned int, 1, scalar_t);
          break;
        case 2:
          RUN(unsigned int, 2, scalar_t);
          break;
        case 3:
          RUN(unsigned int, 3, scalar_t);
          break;
        default:
          RUN(unsigned int, -1, scalar_t);
          break;
      }
    } else {
      TensorInfo<scalar_t, uint64_t> tensorInfo =
        getTensorInfo<scalar_t, THCTensor, uint64_t>(state, tensor);
      TensorInfo<scalar_t, uint64_t> srcInfo =
        getTensorInfo<scalar_t, THCTensor, uint64_t>(state, src);
      TensorInfo<int64_t, uint64_t> indexInfo =
        getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, index);

      RUN(uint64_t, -1, scalar_t)
    }
  }

  if (oldTensor) {
    THCTensor_copyIgnoringOverlaps<scalar_t>(state, oldTensor, tensor);
    THCTensor_(free)(state, tensor);
    tensor = oldTensor;
  }
  THCudaCheck(cudaGetLastError());
}

#undef RUN

#define RUN(TYPE, DIMS, REAL)                                           \
  THCudaTensor_scatterFillKernel<TYPE, REAL, DIMS>                           \
      <<<grid, block, 0, THCState_getCurrentStreamOnDevice(state, curDevice)>>>(      \
          tensorInfo, indexInfo, value, dim, (TYPE)totalElements);

void
THCTensor_(scatterFill)(THCState* state, THCTensor *tensor,
                         int dim, THCudaLongTensor *index, scalar_t value) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, tensor));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  int index_ndim_legacy_all = THCudaLongTensor_nDimensionLegacyAll(state, index);
  THArgCheck(dim >= 0 && dim < THCTensor_(nDimensionLegacyNoScalars)(state, tensor), 2,
             "Index dimension is out of bounds");
  THArgCheck(index_ndim_legacy_all == 0
             || THCudaLongTensor_nDimensionLegacyNoScalars(state, index) == THCTensor_(nDimensionLegacyNoScalars)(state, tensor), 3,
             "Index tensor must be either empty or have same dimensions as output tensor");

  // no-op if index is empty
  if (index_ndim_legacy_all == 0)
      return;

  for (int d = 0; d < THCTensor_(nDimensionLegacyNoScalars)(state, tensor); d++) {
    if (d != dim) {
      THArgCheck(THCTensor_(sizeLegacyNoScalars)(state, tensor, d) ==
                 THCudaLongTensor_sizeLegacyNoScalars(state, index, d), 4,
                 "Index tensor must have same size as output tensor apart from the specified dimension");
    }
  }

  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, tensor) <= MAX_CUTORCH_DIMS,
             1, CUTORCH_DIM_WARNING);

  const ptrdiff_t totalElements = THCudaLongTensor_nElement(state, index);
  const dim3 block = getApplyBlock();
  dim3 grid;
  int curDevice = -1;
  cudaGetDevice(&curDevice);
  THArgCheck(getApplyGrid(state, totalElements, grid, curDevice), 1, CUTORCH_DIM_WARNING);

  THCTensor* oldTensor = NULL;
  if (THCTensor_maybeOverlappingIndices(state, tensor)) {
    oldTensor = tensor;
    tensor = THCTensor_(newContiguous)(state, tensor);
  }

  if (THCTensor_canUse32BitIndexMath(state, tensor) &&
      THCTensor_canUse32BitIndexMath(state, index)) {
    TensorInfo<scalar_t, unsigned int> tensorInfo =
      getTensorInfo<scalar_t, THCTensor, unsigned int>(state, tensor);
    TensorInfo<int64_t, unsigned int> indexInfo =
      getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, index);

    // Specialize for a small number of dimensions.
    switch (indexInfo.dims) {
      case 1:
        RUN(unsigned int, 1, scalar_t);
        break;
      case 2:
        RUN(unsigned int, 2, scalar_t);
        break;
      case 3:
        RUN(unsigned int, 3, scalar_t);
        break;
      default:
        RUN(unsigned int, -1, scalar_t);
        break;
    }
  } else {
    TensorInfo<scalar_t, uint64_t> tensorInfo =
      getTensorInfo<scalar_t, THCTensor, uint64_t>(state, tensor);
    TensorInfo<int64_t, uint64_t> indexInfo =
      getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, index);

    RUN(uint64_t, -1, scalar_t);
  }

  if (oldTensor) {
    THCTensor_copyIgnoringOverlaps<scalar_t>(state, oldTensor, tensor);
    THCTensor_(free)(state, tensor);
    tensor = oldTensor;
  }
  THCudaCheck(cudaGetLastError());
}

#undef RUN

#endif
